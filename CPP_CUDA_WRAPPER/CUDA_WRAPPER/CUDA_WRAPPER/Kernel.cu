﻿

#include "Kernel.cuh"

#include "Cuda_Kernel.cuh"

void Cuda_Kernel::kernel_double(double* A, double* B, double* C, const std::size_t& array_size)
{
	//kernel<double>(A, B, C, array_size);

    double* d_A, * d_B, * d_C;


    hipMalloc((void**)&d_A, array_size * sizeof(double));
    hipMalloc((void**)&d_B, array_size * sizeof(double));
    hipMalloc((void**)&d_C, array_size * sizeof(double));


    hipMemcpy(d_A, A, array_size * sizeof(double), HostToDevice);
    hipMemcpy(d_B, B, array_size * sizeof(double), DeviceToHost);


    dim3 blockSize(512, 1, 1);
    dim3 gridSize(512 / array_size + 1, 1);

    vector_addition_kernel<double> << <1, 1 >> > (d_A, d_B, d_C, array_size);


    hipMemcpy(C, d_C, array_size * sizeof(double), DeviceToHost);
}
