#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <iostream>
#include <random>
#include <memory>
#include <algorithm>
#include <vector>


#define HostToDevice	hipMemcpyHostToDevice
#define DeviceToHost	hipMemcpyDeviceToHost

//constexpr std::size_t Dim_X = 1llu << 9llu;
//constexpr std::size_t Dim_Y = 1llu << 9llu;


//constexpr std::size_t Dim_X = 1 << 10llu;
constexpr std::size_t Dim_X = 5llu;
//constexpr std::size_t Dim_Y = 1 << 10llu;
constexpr std::size_t Dim_Y = 5llu;


//CPU FUNCTIONS
void Print_Matrix(const std::unique_ptr<std::unique_ptr<int[]>[]>& Matrix);

//GPU FUNCTIONS
__global__ void Show_Matrix_GPU(const int* const Matrix);
__global__ void Matrix_Transposition(const int* const Matrix_In, int* const Matrix_Out);

int main(int argc, char* argv[])
{
	int nDevices{};
	int id = hipGetDevice(&id);

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++)
	{
		hipDeviceProp_t prop{};
		hipGetDeviceProperties(&prop, i);
		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem:  %ld\n", static_cast<long>(prop.totalGlobalMem));
		printf("Total constant Mem:  %ld\n", static_cast<long>(prop.totalConstMem));
		printf("Max mem pitch:  %ld\n", static_cast<long>(prop.memPitch));
		printf("Texture Alignment:  %ld\n", static_cast<long>(prop.textureAlignment));

		printf("   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp:  %ld\n", static_cast<long>(prop.sharedMemPerBlock));
		printf("Registers per mp:  %d\n", prop.regsPerBlock);
		printf("Threads in warp:  %d\n", prop.warpSize);
		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf_s("\n");
	}
	printf("%s Starting...\n\n", argv[0]);
	system("pause");

	//CPU
	using type = int;
	using ptr_type = std::unique_ptr<type*>;

	const std::unique_ptr<std::unique_ptr<type[]>[]> Matrix_A(new std::unique_ptr<type[]>[Dim_Y]);
	const std::unique_ptr<std::unique_ptr<type[]>[]> Matrix_B(new std::unique_ptr<type[]>[Dim_Y]);


	for (std::size_t i = 0ull; i < Dim_Y; ++i)
	{
		Matrix_A[i] = std::make_unique<type[]>(Dim_X);
		Matrix_B[i] = std::make_unique<type[]>(Dim_X);

		for (std::size_t j = 0ull; j < Dim_X; ++j)
		{
			Matrix_A[i][j] = static_cast<type>(i * Dim_Y + j + 1);
		}
	}

	//GPU
	type* Dev_Matrix_A{};
	hipMalloc(reinterpret_cast<void**>(&Dev_Matrix_A), Dim_X * Dim_Y * sizeof(type));	//GPU interprets 2D array as a flat array !
	type* Dev_Matrix_B{};
	hipMalloc(reinterpret_cast<void**>(&Dev_Matrix_B), Dim_X * Dim_Y * sizeof(type));	//GPU interprets 2D array as a flat array !

	//Copy memory from CPU to GPU
	for (std::size_t i = 0ull; i < Dim_Y; ++i)
	{
		hipMemcpy(reinterpret_cast<void*>(Dev_Matrix_A + i * Dim_Y), reinterpret_cast<const void*>((Matrix_A.get() + i)->get()), sizeof(type) * Dim_Y, HostToDevice);
	}

	{
		dim3 threads(Dim_X, Dim_Y);
		dim3 blocks(1);
		Matrix_Transposition << <blocks, threads >> > (Dev_Matrix_A, Dev_Matrix_B);
		hipDeviceSynchronize();
	}

	//copying data from GPU to CPU
	for (std::size_t i = 0ull; i < Dim_Y; ++i)
	{
		hipMemcpy(reinterpret_cast<void*>((Matrix_B.get() + i)->get()), reinterpret_cast<const void*>(Dev_Matrix_B + i * Dim_Y), sizeof(type) * Dim_Y, DeviceToHost);
	}

	std::cout << "\nBefore:\n";
	Print_Matrix(Matrix_A);
	std::cout << "After:\n";
	Print_Matrix(Matrix_B);

	hipFree(Dev_Matrix_A);
	hipFree(Dev_Matrix_B);

	system("pause");
	return EXIT_SUCCESS;
}


//DEFINITIONS OF FUNCTIONS


//CPU
void Print_Matrix(const std::unique_ptr<std::unique_ptr<int[]>[]>& Matrix)
{
	for (std::size_t i = 0ull; i < Dim_Y; ++i)
	{
		for (std::size_t j = 0ull; j < Dim_X; ++j)
		{
			std::cout << Matrix[i][j] << ' ';
		}
		std::cout << '\n';
	}
	std::cout << '\n';
}



//GPU
__global__ void Show_Matrix_GPU(const int* const Matrix)
{
	const unsigned int id_x = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int id_y = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int index = id_x + id_y * gridDim.x * gridDim.y;
	const unsigned int threads_amount = blockDim.x * gridDim.x * blockDim.y * gridDim.y;

	//printf("Threads amount %d ", threads_amount);

	while (index < Dim_X * Dim_Y)
	{
		printf("Thread idx: %d | Thread idy %d | Thread id: %d | Value: %d \n", id_x, id_y, index, Matrix[index]);
		index += threads_amount;
	}
	printf("Thread id: %d |\n", index);
}

__global__ void Matrix_Transposition(const int* const Matrix_In, int* const Matrix_Out)
{
	__shared__ int Shared_Matrix_In[Dim_Y][Dim_X];

	const unsigned local_index_x = threadIdx.x;
	const unsigned local_index_y = threadIdx.y;

	const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned int idy = threadIdx.y + blockIdx.y * blockDim.y;
	const unsigned int global_index = idx + idy * blockDim.x;
	const unsigned int global_index_transpose = idy + idx * blockDim.x;

	Shared_Matrix_In[local_index_y][local_index_x] = Matrix_In[global_index];
	__syncthreads();

	Matrix_Out[global_index_transpose] = Shared_Matrix_In[local_index_y][local_index_x];
}