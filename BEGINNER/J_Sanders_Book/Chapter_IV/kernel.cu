#include "hip/hip_runtime.h"
/*
 *		Copyright (c) by Damian Andrysiak. All rights reserved.
*/

#include "hip/hip_runtime.h"
#include ""
#include "../../common/book.h"

#include <stdio.h>
#include <stdlib.h>

#define HostToDevice	hipMemcpyHostToDevice
#define DeviceToHost	hipMemcpyDeviceToHost
#define N 10



__global__ void Add(const int const* a, const int const* b, int* c);

int main(int argc, char* argv[])
{
	int a[N], b[N], c[N];
	int* dev_a, * dev_b, * dev_c;
	// Alokacja pami�ci na GPU
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));
	//Zape�nienie tablic a i b na CPU

	for (int i = 0; i < N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}
	//Kopiowanie tablic a i b do GPU

	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), HostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), HostToDevice));
	Add << <N, 1 >> > (dev_a, dev_b, dev_c);

	// Kopiowanie tablicy c z GPU do CPU
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), DeviceToHost));


	// Wy�wietlenie wyniku
	for (int i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	// Zwolnienie pami�ci alokowanej na GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return EXIT_SUCCESS;
}

__global__ void Add(const int const* a, const int const* b, int* c)
{
	int tid = blockIdx.x;
	if (tid < N)
	{
		c[tid] = a[tid] + b[tid];
	}
}