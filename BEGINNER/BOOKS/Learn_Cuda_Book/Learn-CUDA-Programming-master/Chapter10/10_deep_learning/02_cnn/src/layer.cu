#include "hip/hip_runtime.h"
#include "layer.h"

#include <random>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <cassert>
#include <math.h>
#include <algorithm>

#include <sstream>
#include <fstream>
#include <iostream>

using namespace cudl;

/****************************************************************
 * Layer definition                                             *
 ****************************************************************/
Layer::Layer()
{
	/* do nothing */
}

Layer::~Layer()
{
#if (DEBUG_FORWARD > 0 || DEBUG_BACKWARD > 0)
	std::cout << "Destroy Layer: " << name_ << std::endl;
#endif

	if (output_       != nullptr) { delete output_;       output_       = nullptr; }
	if (grad_input_   != nullptr) { delete grad_input_;   grad_input_   = nullptr; }

	if (weights_      != nullptr) { delete weights_;      weights_	    = nullptr; }
	if (biases_       != nullptr) { delete biases_;	      biases_       = nullptr; }
	if (grad_weights_ != nullptr) { delete grad_weights_; grad_weights_ = nullptr; }
	if (grad_biases_  != nullptr) { delete grad_biases_;  grad_biases_  = nullptr; }
}

void Layer::init_weight_bias(unsigned int seed)
{
	checkCudaErrors(hipDeviceSynchronize());

	if (weights_ == nullptr || biases_ == nullptr)
		return;

	// Create random network
	std::random_device rd;
	std::mt19937 gen(seed == 0 ? rd() : static_cast<unsigned int>(seed));

	// He uniform distribution
	float range = sqrt(6.f / input_->size());	// He's initialization
	std::uniform_real_distribution<> dis(-range, range);

	for (int i = 0; i < weights_->len(); i++)
		weights_->ptr()[i] = static_cast<float>(dis(gen));
	for (int i = 0; i < biases_->len(); i++)
		biases_->ptr()[i] = 0.f;

	// copy initialized value to the device
	weights_->to(DeviceType::cuda);
	biases_->to(DeviceType::cuda);

	std::cout << ".. initialized " << name_ << " layer .." << std::endl;
}

void Layer::update_weights_biases(float learning_rate)
{
	float eps = -1.f * learning_rate;
	if (weights_ != nullptr && grad_weights_ != nullptr)
	{
#if (DEBUG_UPDATE)
		weights_->print(name_ + "::weights (before update)", true);
		grad_weights_->print(name_ + "::gweights", true);
#endif // DEBUG_UPDATE

		// w = w + eps * dw
		checkCublasErrors(
			hipblasSaxpy(cuda_->cublas(),
				weights_->len(),
				&eps,
				grad_weights_->cuda(), 1,
				weights_->cuda(), 1));

#if (DEBUG_UPDATE)
		weights_->print(name_ + "weights (after update)", true);
		// getchar();
#endif // DEBUG_UPDATE
	}

	if (biases_ != nullptr && grad_biases_ != nullptr)
	{
#if (DEBUG_UPDATE)
		biases_->print(name_ + "biases (before update)", true);
		grad_biases_->print(name_ + "gbiases", true);
#endif // DEBUG_UPDATE

		// b = b + eps * db
		checkCublasErrors(
			hipblasSaxpy(cuda_->cublas(),
				biases_->len(),
				&eps,
				grad_biases_->cuda(), 1,
				biases_->cuda(), 1));

#if (DEBUG_UPDATE)
		biases_->print(name_ + "biases (after update)", true);
		// getchar();
#endif // DEBUG_UPDATE
	}
}

float Layer::get_loss(Blob<float> *target)
{
	assert("No Loss layer has no loss." && false);
	return EXIT_FAILURE;
}

int Layer::get_accuracy(Blob<float> *target)
{
	assert("No Loss layer cannot estimate accuracy." && false);
	return EXIT_FAILURE;
}

int Layer::load_parameter()
{
	std::stringstream filename_weights, filename_biases;

	// load weights and biases pretrained parameters
	filename_weights << name_ << ".bin";
	if (weights_->file_read(filename_weights.str()))
		return -1;

	filename_biases << name_ << ".bias.bin";
	if (biases_->file_read(filename_biases.str()))
		return -2;

	std::cout << ".. loaded " << name_ << " pretrain parameter.." << std::endl;

	return 0;
}

int Layer::save_parameter()
{
	std::stringstream filename_weights, filename_biases;

	std::cout << ".. saving " << name_ << " parameter ..";
	
	// Write weights file
	if (weights_)
	{
		filename_weights << name_ << ".bin";
		if (weights_->file_write(filename_weights.str()))
			return -1;
	}
	
	// Write bias file
	if (biases_)
	{
		filename_biases << name_ << ".bias.bin";
		if (biases_->file_write(filename_biases.str()))
			return -2;
	}

	std::cout << " done .." << std::endl;

	return 0;
}

/****************************************************************
 * Dense Layer                                                  *
 ****************************************************************/

Dense::Dense(std::string name, int output_size)
{
	name_ = name;
	output_size_ = output_size;
}

Dense::~Dense()
{
	if (d_one_vec != nullptr) { hipFree(d_one_vec); d_one_vec = nullptr; }
}

__global__ void init_one_vec(float* d_one_vec, size_t length)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= length) return;

	d_one_vec[i] = 1.f;
}

void Dense::fwd_initialize(Blob<float> *input)
{
	// initialize weights and biases
	if (weights_ == nullptr)
	{
		// setup parameter size information
		input_size_  = input->c() * input->h() * input->w();
		
		// initialize weight, bias, and output
		weights_ = new Blob<float>(1, 1, input_size_, output_size_);
		biases_  = new Blob<float>(1, 1, output_size_);
	}

	// initilaize input and output
	if (input_ == nullptr || batch_size_ != input->n())
	{
		input_ = input;
		batch_size_  = input->n();

		if (output_ == nullptr)
			output_  = new Blob<float>(batch_size_, output_size_);
		else
			output_->reset(batch_size_, output_size_);
		
		output_->tensor();

		if (d_one_vec != nullptr)
			hipFree(d_one_vec);
		checkCudaErrors(hipMalloc((void**)&d_one_vec, sizeof(float) * batch_size_));
		init_one_vec<<< (batch_size_+BLOCK_DIM_1D-1)/BLOCK_DIM_1D, BLOCK_DIM_1D >>>(d_one_vec, batch_size_);

		// initialize weights and biases
		if (load_pretrain_ && !freeze_)
		{
			if (load_parameter())
			{
				std::cout << "error occurred.." << std::endl;
				exit(-1);
			}
		}
		else if (!freeze_)
		{
			init_weight_bias();
		}
		else
		{
			/* do nothing */
		}
	}
}

Blob<float> *Dense::forward(Blob<float> *input)
{
	// output = weights^T * input (without biases)
	checkCublasErrors(
		hipblasSgemm(cuda_->cublas(),
			HIPBLAS_OP_T, HIPBLAS_OP_N, 
			output_size_, batch_size_, input_size_,
			&cuda_->one,  
			weights_->cuda(), input_size_, 
			input_->cuda(), input_size_,
			&cuda_->zero, 
			output_->cuda(),  output_size_));

	// output += biases * d_one_vec^T
	checkCublasErrors(hipblasSgemm(cuda_->cublas(),
					HIPBLAS_OP_N, HIPBLAS_OP_N, 
					output_size_, batch_size_, 1,
					&cuda_->one, 
					biases_->cuda(), output_size_, 
					d_one_vec, 1, 
					&cuda_->one, 
					output_->cuda(), output_size_));

#if (DEBUG_DENSE & 0x01)
	input_->print(  name_ + "::input",  true);
	weights_->print(name_ + "::weight", true);
	biases_->print( name_ + "::bias",   true);
	output_->print( name_ + "::output", true);
#endif // DEBUG_DENSE

	return output_;
}

void Dense::bwd_initialize(Blob<float> *grad_output)
{
	if (grad_weights_ == nullptr)
	{
		grad_weights_ = new Blob<float>(weights_->shape());
		grad_biases_  = new Blob<float>(biases_->shape());
	}

	if (grad_input_ == nullptr || batch_size_ != grad_output->n())
	{
		grad_output_  = grad_output;

		if (grad_input_ == nullptr)
			grad_input_   = new Blob<float>(input_->shape());
		else
			grad_input_->reset(input_->shape());
	}
}

Blob<float> *Dense::backward(Blob<float> *grad_output)
{
	// db = (dy) * d_one_vec
	hipblasSgemv(cuda_->cublas(),
			HIPBLAS_OP_N,
			output_size_, batch_size_,
			&cuda_->one,
			grad_output_->cuda(), output_size_,
			d_one_vec, 1,
			&cuda_->zero,
			grad_biases_->cuda(), 1);

	// dw = x * (dy)^T
	hipblasSgemm(cuda_->cublas(),
		HIPBLAS_OP_N, HIPBLAS_OP_T,
		input_size_, output_size_, batch_size_,
		&cuda_->one,
		input_->cuda(),        input_size_,
		grad_output_->cuda(),  output_size_,
		&cuda_->zero,
		grad_weights_->cuda(), input_size_);

	// dx = W * dy
	if (!gradient_stop_)
		hipblasSgemm(cuda_->cublas(),
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			input_size_, batch_size_, output_size_,
			&cuda_->one,
			weights_->cuda(),     input_size_,
			grad_output_->cuda(), output_size_,
			&cuda_->zero, 
			grad_input_->cuda(),  input_size_);

#if (DEBUG_DENSE & 0x02)
	std::cout << name_ << "[BACKWARD]" << std::endl;
	grad_output->print(  name_ + "::gradients", true, grad_output->n());
	grad_weights_->print(name_ + "::gfilter", true);
	grad_biases_->print( name_ + "::gbias", true);
	if (!gradient_stop_)
		grad_input_->print(  name_ + "::gdata", true);
#endif // DEBUG_DENSE

	return grad_input_;
}

/****************************************************************
 * Activation Layer                                             *
 ****************************************************************/

Activation::Activation(std::string name, hipdnnActivationMode_t mode, float coef)
{
	name_ = name;
	act_mode_ = mode;
	act_coef_ = coef;

	hipdnnCreateActivationDescriptor(&act_desc_);
	hipdnnSetActivationDescriptor(act_desc_, act_mode_, HIPDNN_PROPAGATE_NAN, act_coef_);
}

Activation::~Activation()
{
	hipdnnDestroyActivationDescriptor(act_desc_);
}

void Activation::fwd_initialize(Blob<float> *input)
{
	if (input_ == nullptr || batch_size_ != input->n())
	{
		input_ = input;
		input_desc_ = input->tensor();
		batch_size_  = input->n();

		if (output_ == nullptr)
			output_ = new Blob<float>(input->shape());
		else
			output_->reset(input->shape());

		output_desc_ = output_->tensor();
	}
}

Blob<float> *Activation::forward(Blob<float> *input)
{
	hipdnnActivationForward(cuda_->cudnn(),
		act_desc_,
		&cuda_->one,
		input_desc_,
		input->cuda(),
		&cuda_->zero,
		output_desc_,
		output_->cuda());

	return output_;
}

void Activation::bwd_initialize(Blob<float> *grad_output)
{
	if (grad_input_ == nullptr || batch_size_ != grad_output->n())
	{
		grad_output_ = grad_output;

		if (grad_input_ == nullptr)
			grad_input_ = new Blob<float>(input_->shape());
		else
			grad_input_->reset(input_->shape());
	}
}

Blob<float> *Activation::backward(Blob<float> *grad_output)
{
	hipdnnActivationBackward(cuda_->cudnn(),
		act_desc_,
		&cuda_->one, 
		output_desc_, output_->cuda(), 
		output_desc_, grad_output->cuda(), 
		input_desc_, input_->cuda(), 
		&cuda_->zero, 
		input_desc_, grad_input_->cuda());

	return grad_input_;
}

/****************************************************************
 * Softmax definition                                           *
 ****************************************************************/

Softmax::Softmax(std::string name)
{
	name_ = name;
}

Softmax::~Softmax()
{
	// do nothing
}

void Softmax::fwd_initialize(Blob<float> *input)
{
	if (input_ == nullptr || batch_size_ != input->n())
	{
		input_ = input;
		input_desc_ = input->tensor();
		batch_size_  = input->n();
		
		if (output_ == nullptr)
			output_ = new Blob<float>(input->shape());
		else
			output_->reset(input->shape());		

		output_desc_ = output_->tensor();
	}
}

Blob<float> *Softmax::forward(Blob<float> *input)
{
#if (DEBUG_SOFTMAX & 0x01)
	std::cout << name_ << "[FORWARD]" << std::endl;
	input_->print(name_ + "::input", true, input->n());
#endif

	checkCudnnErrors(
		hipdnnSoftmaxForward(cuda_->cudnn(), HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
			&cuda_->one,  input_desc_,  input->cuda(),
			&cuda_->zero, output_desc_, output_->cuda()));

#if (DEBUG_SOFTMAX & 0x01)
	output_->print(name_ + "::output", true, input->n());
#endif

	return output_;
}

void Softmax::bwd_initialize(Blob<float> *target)
{
	if (grad_input_ == nullptr || batch_size_ != target->n())
	{
		if (grad_input_ == nullptr)
			grad_input_ = new Blob<float>(input_->shape());
		else
		 	grad_input_->reset(input_->shape());
	}
}

Blob<float> *Softmax::backward(Blob<float> *target)
{
	// set grad_input_ as predict
	checkCudaErrors(hipMemcpyAsync(grad_input_->cuda(), 
		output_->cuda(), output_->buf_size(), 
		hipMemcpyDeviceToDevice));
	// set grad_input_ = predict - target	
	checkCublasErrors(
		hipblasSaxpy(cuda_->cublas(), target->len(),
			&cuda_->minus_one, target->cuda(), 1,
			grad_input_->cuda(), 1));

	// normalize the grad_output by the batch size
	int grad_output_size = target->n() * target->c() * target->h() * target->w();
	float scale = 1.f / static_cast<float>(target->n());
	checkCublasErrors(hipblasSscal(cuda_->cublas(), grad_output_size, &scale, grad_input_->cuda(), 1));

#if (DEBUG_SOFTMAX & 0x02)
	std::cout << name_ << "[BACKWARD]" << std::endl;
	input_->print( name_ + "::input", true);
	output_->print(name_ + "::predict", true);
	target->print( name_ + "::y", true, target->n());
	grad_input_->print(name_ + "::dx", true, target->n());
#endif

	return grad_input_;
}

float Softmax::get_loss(Blob<float> *target)
{
	return loss_.loss(output_, target);
}

int Softmax::get_accuracy(Blob<float> *target)
{
	int batch_size = output_->n();
	int output_size = output_->size();

	assert(batch_size == target->n());
	assert(output_size == target->size());

	float *h_output, *h_target;
	int idx_output, idx_target;
	int hit_count = 0;

	// get predicts and targets
	h_output = output_->to(host);
	h_target = target->to(host);

	// idx_output = idx_target = 0;
	for (int b = 0; b < batch_size; b++)
	{
		idx_output = 0;
		idx_target = 0;

		for (int i = 1; i < 10; i++)
		{
			if (h_output[b * output_size + i] > h_output[b * output_size + idx_output])
				idx_output = i;
			if (h_target[b * output_size + i] > h_target[b * output_size + idx_target])
				idx_target = i;
		}

		if (idx_output == idx_target)
			hit_count++;
	}

	return hit_count;
}

/****************************************************************
 * Layer definition                                             *
 ****************************************************************/

/**
 * Convolutional layer with bias
 */
Conv2D::Conv2D(std::string name,
				int out_channels, 
				int kernel_size,
				int stride, 
				int padding, 
				int dilation):
				out_channels_(out_channels),
				kernel_size_(kernel_size),
				stride_(stride),
				padding_(padding),
				dilation_(dilation)
{
	name_ = name;

	// create cudnn container handles
	hipdnnCreateFilterDescriptor(&filter_desc_);

	hipdnnCreateConvolutionDescriptor(&conv_desc_);
	checkCudnnErrors(hipdnnSetConvolution2dDescriptor(conv_desc_,
		padding_, padding_, stride_,  stride_, dilation_, dilation_,
		HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

	// setting cudnn convolution math type
	// HIPDNN_DEFAULT_MATH operates convolution with FP32.
	// If you use A100, CUDNN utilise tensor cores with TF32.
	checkCudnnErrors(hipdnnSetConvolutionMathType(conv_desc_, HIPDNN_DEFAULT_MATH));

	d_workspace_ = nullptr;
}

Conv2D::~Conv2D()
{
	// distroy cudnn container resources
	hipdnnDestroyFilterDescriptor(filter_desc_);
	hipdnnDestroyConvolutionDescriptor(conv_desc_);

	// terminate internal created blobs
	if (d_workspace_ != nullptr)	{ hipFree(d_workspace_);	d_workspace_ = nullptr; }
}

void Conv2D::set_workspace()
{
	size_t temp_size = 0;

	// forward
#if CUDNN_MAJOR >= 7
	std::vector<hipdnnConvolutionFwdAlgoPerf_t> 		 fwd_algo_perf_results(HIPDNN_CONVOLUTION_FWD_ALGO_COUNT);
	std::vector<hipdnnConvolutionBwdFilterAlgoPerf_t> bwd_filter_algo_perf_results(HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT);
	std::vector<hipdnnConvolutionBwdDataAlgoPerf_t>	 bwd_data_algo_perf_results(HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM);

	int algo_max_count;
	int returnedAlgoCount = 0;
	checkCudnnErrors(cudnnGetConvolutionForwardAlgorithmMaxCount(cuda_->cudnn(), &algo_max_count));
#if (DEBUG_FIND_ALGO & 1)
	std::cout << this->name_ << ": Available Algorithm Count [FWD]: " << algo_max_count << std::endl;
	checkCudnnErrors(hipdnnFindConvolutionForwardAlgorithm(cuda_->cudnn(),
		input_desc_, filter_desc_, conv_desc_, output_desc_,
		algo_max_count, &returnedAlgoCount, &fwd_algo_perf_results[0]));
	std::cout << "returned algo_count: " << returnedAlgoCount << std::endl;
	for (int i = 0; i < returnedAlgoCount; i++)
		std::cout << "fwd algo[" << i << "] time: " << fwd_algo_perf_results[i].time << ", memory: " << fwd_algo_perf_results[i].memory << std::endl;
#else
	checkCudnnErrors(cudnnGetConvolutionForwardAlgorithm_v7(cuda_->cudnn(),
		input_desc_, filter_desc_, conv_desc_, output_desc_,
		algo_max_count, &returnedAlgoCount, &fwd_algo_perf_results[0]));
#endif
	// shoose the fastest algorithm
	conv_fwd_algo_ = fwd_algo_perf_results[0].algo;
#else
	checkCudnnErrors(hipdnnGetConvolutionForwardAlgorithm(cuda_->cudnn(),
		input_desc_, filter_desc_, conv_desc_, output_desc_,
		HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &conv_fwd_algo_));
#endif
	checkCudnnErrors(hipdnnGetConvolutionForwardWorkspaceSize(cuda_->cudnn(),
		input_desc_, filter_desc_, conv_desc_, output_desc_,
		conv_fwd_algo_, &temp_size));
	workspace_size_ = std::max(workspace_size_, temp_size);

	// bwd - filter
#if CUDNN_MAJOR >= 7
	checkCudnnErrors(cudnnGetConvolutionBackwardFilterAlgorithmMaxCount(cuda_->cudnn(), &algo_max_count));
#if (DEBUG_FIND_ALGO & 1)
	std::cout << this->name_ << ": Available Algorithm Count [BWD-filter]: " << algo_max_count << std::endl;
	checkCudnnErrors(hipdnnFindConvolutionBackwardFilterAlgorithm(cuda_->cudnn(),
		input_desc_, output_desc_, conv_desc_, filter_desc_,
		algo_max_count, &returnedAlgoCount, &bwd_filter_algo_perf_results[0]));
	for (int i = 0; i < returnedAlgoCount; i++)
		std::cout << "bwd filter algo[" << i << "] time: " << fwd_algo_perf_results[i].time << ", memory: " << fwd_algo_perf_results[i].memory << std::endl;
#else
	checkCudnnErrors(cudnnGetConvolutionBackwardFilterAlgorithm_v7(cuda_->cudnn(),
		input_desc_, output_desc_, conv_desc_, filter_desc_,
		algo_max_count, &returnedAlgoCount, &bwd_filter_algo_perf_results[0]));
#endif
	conv_bwd_filter_algo_ = bwd_filter_algo_perf_results[0].algo;
#else
	checkCudnnErrors(hipdnnGetConvolutionBackwardFilterAlgorithm(cuda_->cudnn(),
		input_desc_, output_desc_, conv_desc_, filter_desc_,
		HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &conv_bwd_filter_algo_));
#endif
	checkCudnnErrors(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cuda_->cudnn(),
		input_desc_, output_desc_, conv_desc_, filter_desc_,
		conv_bwd_filter_algo_, &temp_size));
	workspace_size_ = std::max(workspace_size_, temp_size);

	// bwd - data
#if CUDNN_MAJOR >= 7
	checkCudnnErrors(cudnnGetConvolutionBackwardDataAlgorithmMaxCount(cuda_->cudnn(), &algo_max_count));
#if (DEBUG_FIND_ALGO & 1)
	std::cout << this->name_ << ": Available Algorithm Count [BWD-data]: " << algo_max_count << std::endl;
	checkCudnnErrors(hipdnnFindConvolutionBackwardDataAlgorithm(cuda_->cudnn(),
		filter_desc_, output_desc_, conv_desc_, input_desc_,
		algo_max_count, &returnedAlgoCount, &bwd_data_algo_perf_results[0]));
	for (int i = 0; i < returnedAlgoCount; i++)
		std::cout << "bwd data algo[" << i << "] time: " << fwd_algo_perf_results[i].time << ", memory: " << fwd_algo_perf_results[i].memory << std::endl;
#else
	checkCudnnErrors(cudnnGetConvolutionBackwardDataAlgorithm_v7(cuda_->cudnn(),
		filter_desc_, output_desc_, conv_desc_, input_desc_,
		algo_max_count, &returnedAlgoCount, &bwd_data_algo_perf_results[0]));
#endif
	conv_bwd_data_algo_ = bwd_data_algo_perf_results[0].algo;
#else
	checkCudnnErrors(hipdnnGetConvolutionBackwardDataAlgorithm(cuda_->cudnn(), 
		filter_desc_, output_desc_, conv_desc_, input_desc_, 
		HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &conv_bwd_data_algo_));
#endif
	checkCudnnErrors(hipdnnGetConvolutionBackwardDataWorkspaceSize(cuda_->cudnn(),
		filter_desc_, output_desc_, conv_desc_, input_desc_,
		conv_bwd_data_algo_, &temp_size));
	workspace_size_ = std::max(workspace_size_, temp_size);

	if (workspace_size_ > 0)
	{
		if (d_workspace_ != nullptr)
			checkCudaErrors(hipFree(d_workspace_));
		checkCudaErrors(hipMalloc((void**)&d_workspace_, workspace_size_));
	}
}

void Conv2D::fwd_initialize(Blob<float> *input)
{
	// initialize weights and bias
	if (weights_ == nullptr)
	{
		// initialize containers handles
		checkCudnnErrors(hipdnnSetFilter4dDescriptor(filter_desc_, 
			HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
			out_channels_, input->c(), kernel_size_, kernel_size_));

		weights_ = new Blob<float>(out_channels_, input->c(), kernel_size_, kernel_size_);
		biases_  = new Blob<float>(1, out_channels_);	// bias size
		bias_desc_ = biases_->tensor();
	}
 
	// initilaize input and output
	if (input_ == nullptr || batch_size_ != input->n())
	{
		// initialize input
		input_ = input;
		input_desc_ = input->tensor();
		batch_size_  = input->n();

		// initilaize output
		checkCudnnErrors(hipdnnGetConvolution2dForwardOutputDim(
			conv_desc_, input_desc_, filter_desc_,
			&output_size_[0], &output_size_[1], &output_size_[2], &output_size_[3]));

		if (output_ == nullptr)
			output_  = new Blob<float>(output_size_);
		else
			output_->reset(output_size_);

		output_desc_ = output_->tensor();

		// initialize workspace for cudnn
		set_workspace();

		// initialize weights
		if (load_pretrain_ && !freeze_)
		{
			if (load_parameter())
			{
				std::cout << "error occurred.." << std::endl;
				exit(-1);
			}
		}
		else if (!freeze_)
		{
			init_weight_bias();
		}
		else
		{
			/* do nothing */
		}
	}
}

Blob<float> *Conv2D::forward(Blob<float> *input)
{
	checkCudnnErrors(hipdnnConvolutionForward(cuda_->cudnn(),
		&cuda_->one,  input_desc_,  input_->cuda(),
		filter_desc_, weights_->cuda(), conv_desc_, conv_fwd_algo_, d_workspace_,  workspace_size_,
		&cuda_->zero, output_desc_, output_->cuda()));

	checkCudnnErrors(hipdnnAddTensor(cuda_->cudnn(), 
		&cuda_->one, bias_desc_, biases_->cuda(), 
		&cuda_->one, output_desc_, output_->cuda()));

#if (DEBUG_CONV & 0x01)
	input_->print(  name_ + "::input", true, input_->n(), 28);
	weights_->print(name_ + "::weight", true);
	biases_->print( name_ + "::bias", true);
	output_->print( name_ + "::output", true);
#endif

	return output_;
}

void Conv2D::bwd_initialize(Blob<float> *grad_output)
{
	if (grad_weights_ == nullptr) {
		grad_weights_ = new Blob<float>(weights_->shape());
		grad_biases_  = new Blob<float>(1, biases_->c());
	}

	// initialize grad_output back-propagation space
	if (grad_input_ == nullptr || batch_size_ != grad_output->n()) {
		grad_output_  = grad_output;

		if (grad_input_ == nullptr)
			grad_input_ = new Blob<float>(input_->shape());
		else
			grad_input_->reset(input_->shape());
	}
}

Blob<float> *Conv2D::backward(Blob<float> *grad_output)
{
	// gradients of biases
	checkCudnnErrors(
		hipdnnConvolutionBackwardBias(cuda_->cudnn(),
			&cuda_->one,  
			output_desc_, grad_output->cuda(),
			&cuda_->zero, 
			bias_desc_,   grad_biases_->cuda()));

	// gradients of weights 
	checkCudnnErrors(
		hipdnnConvolutionBackwardFilter(cuda_->cudnn(),
			&cuda_->one, 
			input_desc_, input_->cuda(), 
			output_desc_, grad_output_->cuda(),
			conv_desc_, conv_bwd_filter_algo_, d_workspace_, workspace_size_,
			&cuda_->zero, 
			filter_desc_, grad_weights_->cuda()));

	// gradients of input data
	if (!gradient_stop_)
		checkCudnnErrors(
			hipdnnConvolutionBackwardData(cuda_->cudnn(),
				&cuda_->one, 
				filter_desc_, weights_->cuda(), 
				output_desc_, grad_output->cuda(), 
				conv_desc_, conv_bwd_data_algo_, d_workspace_, workspace_size_,
				&cuda_->zero, 
				input_desc_, grad_input_->cuda()));

#if (DEBUG_CONV & 0x02)
	std::cout << name_ << "[BACKWARD]" << std::endl;
	grad_output->print( name_ + "::gradients", true);
	grad_biases_->print(name_ + "gbias", true);
	grad_weights_->print(name_+ "gfilter", true);
	if (!gradient_stop_)
		grad_input_->print(name_+"gdata", true);
#endif

#if (DEBUG_CONV & 0x04)
	grad_output->print( name_ + "::gradients", true);
	grad_biases_->print( name_ + "::gbias", true);
#endif

	return grad_input_;
}

/****************************************************************
 * Layer definition                                             *
 ****************************************************************/

Pooling::Pooling(std::string name,
	int kernel_size, 
	int padding, 
	int stride,
	hipdnnPoolingMode_t mode):
		kernel_size_(kernel_size),
		padding_(padding),
		stride_(stride),
			mode_(mode)
{
	name_ = name;

	hipdnnCreatePoolingDescriptor(&pool_desc_);
	hipdnnSetPooling2dDescriptor(pool_desc_, mode_, HIPDNN_PROPAGATE_NAN,
		kernel_size_, kernel_size_, padding_, padding_, stride_, stride_);
}

Pooling::~Pooling()
{
	hipdnnDestroyPoolingDescriptor(pool_desc_);
}

void Pooling::fwd_initialize(Blob<float> *input)
{
	if (input_ == nullptr || batch_size_ != input->n())
	{
		input_ = input;

		// resource initialize
		input_desc_ = input_->tensor();
		batch_size_  = input->n();
		
		// setting output
		hipdnnGetPooling2dForwardOutputDim(pool_desc_, input_desc_, 
			&output_size_[0], &output_size_[1], &output_size_[2], &output_size_[3]);
		if (output_ == nullptr)
			output_ = new Blob<float>(output_size_);
		else
			output_->reset(output_size_);
		
		output_desc_ = output_->tensor();
	}
}

Blob<float> *Pooling::forward(Blob<float> *input)
{
	hipdnnPoolingForward(cuda_->cudnn(), pool_desc_,
		&cuda_->one,   input_desc_,  input_->cuda(),
		&cuda_->zero,  output_desc_, output_->cuda());

	return output_;
}

void Pooling::bwd_initialize(Blob<float> *grad_output)
{
	if (grad_input_ == nullptr || batch_size_ != grad_output->n())
	{
		grad_output_ = grad_output;

		if (grad_input_ == nullptr)
			grad_input_ = new Blob<float>(input_->shape());
		else
			grad_input_->reset(input_->shape());
	}
}

Blob<float> *Pooling::backward(Blob<float> *grad_output)
{
	checkCudnnErrors(
		hipdnnPoolingBackward(cuda_->cudnn(), pool_desc_,
			&cuda_->one,  
			output_desc_, output_->cuda(),
			output_desc_, grad_output->cuda(), 
			input_desc_,  input_->cuda(), 
			&cuda_->zero, 
			input_desc_,  grad_input_->cuda()));

	return grad_input_;
}

