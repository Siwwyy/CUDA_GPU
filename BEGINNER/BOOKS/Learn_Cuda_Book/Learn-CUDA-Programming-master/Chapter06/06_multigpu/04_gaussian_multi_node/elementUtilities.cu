#include "gaussian_multi_gpu_rdma.h"

void resetElement(elemtype* e)
{
    #if (PACK_SIZE == 32)
    {
        *e = 0;
    }
    #elif (PACK_SIZE == 128)
    {
        e->x = 0;
        e->y = 0;
        e->z = 0;
        e->w = 0;
    }
    #else
    {
        #error "Unsupported PACK_SIZE detected"
    }
    #endif
}

void setElementBit(elemtype* e, int bitIndex)
{
    #if (PACK_SIZE == 32)
    {
        (*e) |= (1 << bitIndex);
    }
    #elif (PACK_SIZE == 128)
    {
        if(bitIndex < 32)
        {
            e->x |= (1 << bitIndex);
        }
        else if(bitIndex < 64)
        {
            bitIndex -= 32;
            e->y |= (1 << bitIndex);
        }
        else if(bitIndex < 96)
        {
            bitIndex -= 64;
            e->z |= (1 << bitIndex);
        }
        else
        {
            bitIndex -= 96;
            e->w |= (1 << bitIndex);
        }
    }
    #else
    {
        #error "Unsupported PACK_SIZE detected"
    }
    #endif
}

unsigned char getElementBit(elemtype* e, int bitIndex)
{
    unsigned char bitValue;

    #if (PACK_SIZE == 32)
    {
        bitValue = ((*e) >> bitIndex ) & 1;
    }
    #elif (PACK_SIZE == 128)
    {
        if(bitIndex < 32)
        {
            bitValue = (e->x >> bitIndex) & 1;
        }
        else if(bitIndex < 64)
        {
            bitIndex -= 32;
            bitValue = (e->y >> bitIndex) & 1;
        }
        else if(bitIndex < 96)
        {
            bitIndex -= 64;
            bitValue = (e->z >> bitIndex) & 1;
        }
        else
        {
            bitIndex -= 96;
            bitValue = (e->w >> bitIndex) & 1;
        }
    }
    #else
    {
        #error "Unsupported PACK_SIZE detected"
    }
    #endif
    return bitValue;
}


__device__ unsigned int bfe(unsigned int x, unsigned int bit, unsigned int numBits) 
{
    unsigned int ret;
    asm("bfe.u32 %0, %1, %2, %3;" : "=r"(ret) : "r"(x), "r"(bit), "r"(numBits));
    return ret;
}


__device__ unsigned int bfew(elemtype* e, int bitIndex)
{
    #if (PACK_SIZE == 32)
    {
         return bfe(*e, bitIndex, 1);
    }
    #elif (PACK_SIZE == 128)
    {
        if(bitIndex < 32)
        {
            return bfe(e->x, bitIndex, 1);
        }
        else if(bitIndex < 64)
        {
            bitIndex -= 32;
            return bfe(e->y, bitIndex, 1);
        }
        else if(bitIndex < 96)
        {
            bitIndex -= 64;
            return bfe(e->z, bitIndex, 1);
        }
        else
        {
            bitIndex -= 96;
            return bfe(e->w, bitIndex, 1);
        }
    }
    #else
    {
        #error "Unsupported PACK_SIZE detected"
    }
    #endif
}
